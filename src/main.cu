#include "hip/hip_runtime.h"
/*
  ======================================================================
   demo.c --- protoype to show off the simple solver
  ----------------------------------------------------------------------
   Author : Jos Stam (jstam@aw.sgi.com)
   Creation Date : Jan 9 2003

   Description:

        This code is a simple prototype that demonstrates how to use the
        code provided in my GDC2003 paper entitles "Real-Time Fluid Dynamics
        for Games". This code uses OpenGL and GLUT for graphics and interface

  =======================================================================
*/

#include <sys/time.h>
#include "wtime.h"
#include <stddef.h>
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "indices.h"
#include "solver.h"
#include "wtime.h"

/* macros */

#define IX(x,y) (rb_idx((x),(y),(N+2)))

/* global variables */

static int N;
static float dt, diff, visc;
static float force, source;

static float * h_u, * h_v, * h_u_prev, * h_v_prev;
static float * h_dens, * h_dens_prev;

float * d_u, * d_v, * d_u_prev, * d_v_prev;
float * d_dens, * d_dens_prev;

/*
  ----------------------------------------------------------------------
   free/clear/allocate simulation data
  ----------------------------------------------------------------------
*/


static void free_data ( void )
{
        if ( h_u ) free ( h_u );
        if ( h_v ) free ( h_v );
        if ( h_u_prev ) free ( h_u_prev );
        if ( h_v_prev ) free ( h_v_prev );
        if ( h_dens ) free ( h_dens );
        if ( h_dens_prev ) free ( h_dens_prev );
  if ( d_u ) hipFree ( d_u );
        if ( d_v ) hipFree ( d_v );
        if ( d_u_prev ) hipFree ( d_u_prev );
        if ( d_v_prev ) hipFree ( d_v_prev );
        if ( d_dens ) hipFree ( d_dens );
        if ( d_dens_prev ) hipFree ( d_dens_prev );
}

static void clear_data ( void )
{
        int i, size=(N+2)*(N+2);

        for ( i=0 ; i<size ; i++ ) {
                h_u[i] = h_v[i] = h_u_prev[i] = h_v_prev[i] = h_dens[i] = h_dens_prev[i] = 0.0f;
        }
}

static int allocate_data ( void )
{
        int size = (N+2)*(N+2);

        // h_u                  = (float *) malloc( size*sizeof(float) );
        // h_v                  = (float *) malloc( size*sizeof(float) );
        // h_u_prev             = (float *) malloc( size*sizeof(float) );
        // h_v_prev             = (float *) malloc( size*sizeof(float) );
        // h_dens               = (float *) malloc( size*sizeof(float) );
        // h_dens_prev  = (float *) malloc( size*sizeof(float) );

  h_u = (float*) malloc(size*sizeof(float));
  h_v = (float*) malloc(size*sizeof(float));
  h_u_prev = (float*) malloc(size*sizeof(float));
  h_v_prev = (float*) malloc(size*sizeof(float));
  h_dens = (float*) malloc(size*sizeof(float));
  h_dens_prev = (float*) malloc(size*sizeof(float));

  hipMalloc((void**) &d_u, size*sizeof(float));
  hipMalloc((void**) &d_v, size*sizeof(float));
  hipMalloc((void**) &d_u_prev, size*sizeof(float));
  hipMalloc((void**) &d_v_prev, size*sizeof(float));
  hipMalloc((void**) &d_dens, size*sizeof(float));
  hipMalloc((void**) &d_dens_prev, size*sizeof(float));

        if ( !h_u || !h_v || !h_u_prev || !h_v_prev || !h_dens || !h_dens_prev ) {
                fprintf ( stderr, "cannot allocate data\n" );
                return ( 0 );
        }

        return ( 1 );
}



static void react ( float * d, float * h_u, float * h_v )
{
        int i, size = (N+2)*(N+2);
        float max_velocity2 = 0.0f;
        float max_density = 0.0f;

        max_velocity2 = max_density = 0.0f;
        for ( i=0 ; i<size ; i++ ) {
                if (max_velocity2 < h_u[i]*h_u[i] + h_v[i]*h_v[i]) {
                        max_velocity2 = h_u[i]*h_u[i] + h_v[i]*h_v[i];
                }
                if (max_density < d[i]) {
                        max_density = d[i];
                }
        }

        for ( i=0 ; i<size ; i++ ) {
                h_u[i] = h_v[i] = d[i] = 0.0f;
        }

        if (max_velocity2<0.0000005f) {
                h_u[IX(N/2,N/2)] = force * 10.0f;
                h_v[IX(N/2,N/2)] = force * 10.0f;
        }
        if (max_density<1.0f) {
                d[IX(N/2,N/2)] = source * 10.0f;
        }

        return;
}

static void one_step ( void )
{
        static int times = 1;
        static double start_t = 0.0;
        static double one_second = 0.0;
        static double react_ns_p_cell = 0.0;
        static double vel_ns_p_cell = 0.0;
        static double dens_ns_p_cell = 0.0;

        start_t = wtime();
        react ( h_dens_prev, h_u_prev, h_v_prev );
        react_ns_p_cell += 1.0e9 * (wtime()-start_t)/(N*N);

        start_t = wtime();
        vel_step ( N, h_u, h_v, h_u_prev, h_v_prev, visc, dt );
        vel_ns_p_cell += 1.0e9 * (wtime()-start_t)/(N*N);

        start_t = wtime();
        dens_step ( N, h_dens, h_dens_prev, h_u, h_v, diff, dt );
        dens_ns_p_cell += 1.0e9 * (wtime()-start_t)/(N*N);

        if (1.0<wtime()-one_second) { /* at least 1s between stats */
                printf("%lf, %lf, %lf, %lf: ns per cell total, react, vel_step, dens_step\n",
                        (react_ns_p_cell+vel_ns_p_cell+dens_ns_p_cell)/times,
                        react_ns_p_cell/times, vel_ns_p_cell/times, dens_ns_p_cell/times);
                one_second = wtime();
                react_ns_p_cell = 0.0;
                vel_ns_p_cell = 0.0;
                dens_ns_p_cell = 0.0;
                times = 1;
        } else {
                times++;
        }
}

/*
  ----------------------------------------------------------------------
   main --- main routine
  ----------------------------------------------------------------------
*/

int main ( int argc, char ** argv )
{
        int i = 0;

        if ( argc > 2 && argc != 7 ) {
                fprintf ( stderr, "usage : %s N dt diff visc force source\n", argv[0] );
                fprintf ( stderr, "where:\n" );\
                fprintf ( stderr, "\t N      : grid resolution\n" );
                fprintf ( stderr, "\t dt     : time step\n" );
                fprintf ( stderr, "\t diff   : diffusion rate of the density\n" );
                fprintf ( stderr, "\t visc   : viscosity of the fluid\n" );
                fprintf ( stderr, "\t force  : scales the mouse movement that generate a force\n" );
                fprintf ( stderr, "\t source : amount of density that will be deposited\n" );
                exit ( 1 );
        }

        if ( argc <= 2 ) {
                N = argc == 2 ? atoi(argv[1]) : 64;
                dt = 0.1f;
                diff = 0.0f;
                visc = 0.0f;
                force = 5.0f;
                source = 100.0f;
                fprintf ( stderr, "Using defaults : N=%d dt=%g diff=%g visc=%g force = %g source=%g\n",
                        N, dt, diff, visc, force, source );
        } else {
                N = atoi(argv[1]);
                dt = atof(argv[2]);
                diff = atof(argv[3]);
                visc = atof(argv[4]);
                force = atof(argv[5]);
                source = atof(argv[6]);
        }

        if ( !allocate_data () ) exit ( 1 );
        clear_data ();

  unsigned int size = (N+2)*(N+2);
  hipMemcpy(h_dens, d_dens, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(h_dens_prev, d_dens_prev, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(h_u, d_u, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(h_u_prev, d_u_prev, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(h_v, d_v, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(h_v_prev, d_v_prev, size * sizeof(float), hipMemcpyHostToDevice);
        for (i=0; i<2048; i++) {
    one_step ();
    hipMemcpy(d_dens, h_dens, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(d_dens_prev, h_dens_prev, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(d_u, h_u, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(d_u_prev, h_u_prev, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(d_v, h_v, size * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(d_v_prev, h_v_prev, size * sizeof(float), hipMemcpyDeviceToHost);
  }
  free_data ();

        exit ( 0 );
}
//
// timing.c
//
double wtime(void)
{
  struct timeval tv;
  gettimeofday(&tv, 0);

  return (double) tv.tv_sec + 1e-6 * tv.tv_usec;
}
#undef IX
#define IX(x,y) (rb_idx((x),(y),(n+2)))
#define SWAP(x0,x) {float * tmp=x0;x0=x;x=tmp;}

int threadsPerBlock = 1024;


typedef enum { NONE = 0, VERTICAL = 1, HORIZONTAL = 2 } boundary;
typedef enum { RED, BLACK } grid_color;

__global__ void add_source_kernel(unsigned int n, float *x, const float *s, float dt)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int size = (n + 2) * (n + 2);
  if (i < size) {
    x[i] += dt * s[i];
  }
}

static void add_source(unsigned int n, float *x, const float *s, float dt)
{
  unsigned int size = (n + 2) * (n + 2);
  int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;

  // Check if the number of blocks exceeds the maximum allowed
  int maxBlocks = 65535; // Maximum number of blocks (adjust if needed)
  while (numBlocks > maxBlocks) {
    add_source_kernel<<<maxBlocks, threadsPerBlock>>>(n, x, s, dt);
    x += maxBlocks * threadsPerBlock;
    s += maxBlocks * threadsPerBlock;
    numBlocks -= maxBlocks;
  }
  add_source_kernel<<<numBlocks, threadsPerBlock>>>(n, x, s, dt);
}

__global__ void set_bnd_kernel(unsigned int n, boundary b, float* x)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  // int
  if (i < n + 1) {
    x[IX(0, i)]     = (b == VERTICAL)   ? -x[IX(1, i)] : x[IX(1, i)];
    x[IX(n + 1, i)] = (b == VERTICAL)   ? -x[IX(n, i)] : x[IX(n, i)];
    x[IX(i, 0)]     = (b == HORIZONTAL) ? -x[IX(i, 1)] : x[IX(i, 1)];
    x[IX(i, n + 1)] = (b == HORIZONTAL) ? -x[IX(i, n)] : x[IX(i, n)];
  }
}

static void set_bnd(unsigned int n, boundary b, float* x)
{
    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(n, b, x);
    x[IX(0, 0)] = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
    x[IX(n + 1, 0)] = 0.5f * (x[IX(n, 0)] + x[IX(n + 1, 1)]);
    x[IX(0, n + 1)] = 0.5f * (x[IX(1, n + 1)] + x[IX(0, n)]);
    x[IX(n + 1, n + 1)] = 0.5f * (x[IX(n, n + 1)] + x[IX(n + 1, n)]);
}

__global__ void lin_solve_rb_step(grid_color color,
  unsigned int n,
  float a,
  float c,
  const float * same0,
  const float * neigh,
  float * same)
  {
    unsigned int width = (n + 2) / 2;
    unsigned int block_size = 1024 / n;

    unsigned y = blockIdx.y;
    unsigned x = blockIdx.x * blockDim.x + threadIdx.x;

    int shift = color == RED ? 1 : -1;
    unsigned int start = color == RED ? 0 : 1;

    // for (unsigned int i = 0; i < ((n+2) / NUM_BLOCKS) ; i++) {
    // const float*  same0_i = same0 + (i * block_size);
    // const float*  neigh_i = neigh + (i * block_size);
    // float*  same_i = same + (i * block_size);

    int index = idx(x+start,y+1,width);
    same[index] = (same0[index] + a * (neigh[index - width] +
    neigh[index] +
    neigh[index + (y % 2 == 0 ? shift : -shift)] +
    neigh[index + width])) / c;
    // }
}

void lin_solve(unsigned int n, boundary b,
                      float * x,
                      const float * x0,
                      float a, float c)
{
    unsigned int color_size = (n + 2) * ((n + 2) / 2);
    const float * red0 = x0;
    const float * blk0 = x0 + color_size;
    float * red = x;
    float * blk = x + color_size;

    unsigned int blocksPerRow = (((n / 2) + 1023) / 1024);
    dim3 grid(blocksPerRow, n);
    dim3 block(1024, 1);
    for (unsigned int k = 0; k < 20; ++k) {
        // hipMemcpyToSymbol(HIP_SYMBOL(ro_mem), red0, threadsPerBlock * sizeof(float));
        lin_solve_rb_step<<<grid, block>>>(RED, n, a, c, red0, blk, red);
        // hipMemcpyToSymbol(HIP_SYMBOL(ro_mem), blk0, threadsPerBlock * sizeof(float));
        lin_solve_rb_step<<<grid, block>>>(BLACK, n, a, c, blk0, red, blk);
        set_bnd(n, b, x);
    }
  }

void diffuse(unsigned int n, boundary b, float * x, const float * x0, float diff, float dt)
{
    float a = dt * diff * n * n;
    lin_solve(n, b, x, x0, a, 1 + 4 * a);
}

//float max(float x, float y) {
// return x < y ? y : x;
//}

//float min(float x, float y) {
//  return x < y ? x : y;
//}


__global__ void advect_kernel(unsigned int n, boundary b, float*  d, float* d0, const float* u, const float* v, float dt) {
  float dt0 = dt * n;
  unsigned int i = blockDim.y * blockIdx.y + threadIdx.y + 1;
  unsigned int j = blockDim.x * blockIdx.x + threadIdx.x + 1;
  if (i < n+1 && j < n+1) {
    int i0, i1, j0, j1;
    float x, y, s0, t0, s1, t1;
    x = i - dt0 * u[IX(i, j)];
    y = j - dt0 * v[IX(i, j)];
    x = max(x, 0.5f);
    x = min(x, n + 0.5f);
    i0 = (int)x;
    i1 = i0 + 1;
    y = max(y, 0.5f);
    y = min(y, n + 0.5f);
    j0 = (int)y;
    j1 = j0 + 1;
    s1 = x - i0;
    s0 = 1 - s1;
    t1 = y - j0;
    t0 = 1 - t1;
    d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) + s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
  }
}

void advect(unsigned int n, boundary b, float*  d, float* d0, const float* u, const float* v, float dt)
{
  unsigned int numBlocks = (n + 31) / 32;
  dim3 block(32, 32);
  dim3 grid(numBlocks,numBlocks);
  advect_kernel<<<grid, block>>>(n, b, d, d0, u, v, dt);
  set_bnd(n, b, d);
}

__global__ void project_density_kernel(unsigned int n, float *u, float *v, float *p, float *div) {
  unsigned int i = blockDim.y * blockIdx.y + threadIdx.y + 1;
  unsigned int j = blockDim.x * blockIdx.x + threadIdx.x + 1;
  if (i < n+1 && j < n+1) {
    div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] +
                            v[IX(i, j + 1)] - v[IX(i, j - 1)]) / n;
    p[IX(i, j)] = 0;
  }
}

__global__ void project_vel_kernel(unsigned int n, float *u, float *v, float *p) {
  unsigned int i = blockDim.y * blockIdx.y + threadIdx.y + 1;
  unsigned int j = blockDim.x * blockIdx.x + threadIdx.x + 1;
  if (i < n+1 && j < n+1) {
    u[IX(i, j)] -= 0.5f * n * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
    v[IX(i, j)] -= 0.5f * n * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
  }
}

static void project(unsigned int n, float * u, float *  v, float * p, float * div)
{
        // printf("Thread %d in range [%d,%d), total: %d\n", omp_get_thread_num(), start+1, end+1, n);
    unsigned int numBlocks = (n + 31) / 32;
    dim3 block(32, 32);
    dim3 grid(numBlocks,numBlocks);
    project_density_kernel<<<grid,block>>>(n, u, v, p, div);

    set_bnd(n, NONE, div);
    set_bnd(n, NONE, p);

    lin_solve(n, NONE, p, div, 1, 4);

    project_vel_kernel<<<grid, block>>>(n, u, v, p);
    set_bnd(n, VERTICAL, u);
    set_bnd(n, HORIZONTAL, v);
}

__host__ void dens_step(unsigned int n, float *x, float *x0, float *u, float *v, float diff, float dt)
{
    add_source(n, x, x0, dt);
    SWAP(x0, x);
    diffuse(n, NONE, x, x0, diff, dt);
    SWAP(x0, x);
    advect(n, NONE, x, x0, u, v, dt);
}

__host__ void vel_step(unsigned int n, float *u, float *v, float *u0, float *v0, float visc, float dt)
{
    add_source(n, u, u0, dt);
    add_source(n, v, v0, dt);
    SWAP(u0, u);
    diffuse(n, VERTICAL, u, u0, visc, dt);
    SWAP(v0, v);
    diffuse(n, HORIZONTAL, v, v0, visc, dt);
    project(n, u, v, u0, v0);
    SWAP(u0, u);
    SWAP(v0, v);
    advect(n, VERTICAL, u, u0, u0, v0, dt);
    advect(n, HORIZONTAL, v, v0, u0, v0, dt);
    project(n, u, v, u0, v0);
}
