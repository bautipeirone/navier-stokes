#include "hip/hip_runtime.h"
#include <stddef.h>
#include <hip/hip_runtime.h>

#include "solver.h"
#include "indices.h"
#include <stdio.h>

#define IX(x,y) (rb_idx((x),(y),(n+2)))
#define SWAP(x0,x) {float * tmp=x0;x0=x;x=tmp;}

int threadsPerBlock = 1024;


typedef enum { NONE = 0, VERTICAL = 1, HORIZONTAL = 2 } boundary;
typedef enum { RED, BLACK } grid_color;

__global__ void add_source_kernel(unsigned int n, float *x, const float *s, float dt)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int size = (n + 2) * (n + 2);
  if (i < size) {
    x[i] += dt * s[i];
  }
}

static void add_source(unsigned int n, float *x, const float *s, float dt)
{
  unsigned int size = (n + 2) * (n + 2);
  int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;
  
  // Check if the number of blocks exceeds the maximum allowed
  int maxBlocks = 65535; // Maximum number of blocks (adjust if needed)
  while (numBlocks > maxBlocks) {
    add_source_kernel<<<maxBlocks, threadsPerBlock>>>(n, x, s, dt);
    x += maxBlocks * threadsPerBlock;
    s += maxBlocks * threadsPerBlock;
    numBlocks -= maxBlocks;
  }
  add_source_kernel<<<numBlocks, threadsPerBlock>>>(n, x, s, dt);
}

__global__ void set_bnd_kernel(unsigned int n, boundary b, float* x)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  // int 
  if (i < n + 1) {
    x[IX(0, i)]     = (b == VERTICAL)   ? -x[IX(1, i)] : x[IX(1, i)];
    x[IX(n + 1, i)] = (b == VERTICAL)   ? -x[IX(n, i)] : x[IX(n, i)];
    x[IX(i, 0)]     = (b == HORIZONTAL) ? -x[IX(i, 1)] : x[IX(i, 1)];
    x[IX(i, n + 1)] = (b == HORIZONTAL) ? -x[IX(i, n)] : x[IX(i, n)];
  }
}

static void set_bnd(unsigned int n, boundary b, float* x)
{
    int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    set_bnd_kernel<<<numBlocks, threadsPerBlock>>>(n, b, x);
    x[IX(0, 0)] = 0.5f * (x[IX(1, 0)] + x[IX(0, 1)]);
    x[IX(n + 1, 0)] = 0.5f * (x[IX(n, 0)] + x[IX(n + 1, 1)]);
    x[IX(0, n + 1)] = 0.5f * (x[IX(1, n + 1)] + x[IX(0, n)]);
    x[IX(n + 1, n + 1)] = 0.5f * (x[IX(n, n + 1)] + x[IX(n + 1, n)]);
}

__global__ void lin_solve_rb_step(grid_color color,
  unsigned int n,
  float a,
  float c,
  const float * same0,
  const float * neigh,
  float * same)
  {
    unsigned int width = (n + 2) / 2;
    unsigned int block_size = 1024 / n;

    unsigned y = blockIdx.y;
    unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    
    int shift = color == RED ? 1 : -1;
    unsigned int start = color == RED ? 0 : 1;
    
    // for (unsigned int i = 0; i < ((n+2) / NUM_BLOCKS) ; i++) {
    // const float*  same0_i = same0 + (i * block_size);
    // const float*  neigh_i = neigh + (i * block_size);
    // float*  same_i = same + (i * block_size);             

    int index = idx(x+start,y+1,width);
    same[index] = (same0[index] + a * (neigh[index - width] +
    neigh[index] +
    neigh[index + (y % 2 == 0 ? shift : -shift)] +
    neigh[index + width])) / c;
    // }
}

void lin_solve(unsigned int n, boundary b,
                      float * x,
                      const float * x0,
                      float a, float c)
{
    unsigned int color_size = (n + 2) * ((n + 2) / 2);
    const float * red0 = x0;
    const float * blk0 = x0 + color_size;
    float * red = x;
    float * blk = x + color_size;

    unsigned int blocksPerRow = (((n / 2) + 1023) / 1024);
    dim3 grid(blocksPerRow, n);
    dim3 block(1024, 1);
    for (unsigned int k = 0; k < 20; ++k) {
        // hipMemcpyToSymbol(HIP_SYMBOL(ro_mem), red0, threadsPerBlock * sizeof(float));
        lin_solve_rb_step<<<grid, block>>>(RED, n, a, c, red0, blk, red);
        // hipMemcpyToSymbol(HIP_SYMBOL(ro_mem), blk0, threadsPerBlock * sizeof(float));
        lin_solve_rb_step<<<grid, block>>>(BLACK, n, a, c, blk0, red, blk);
        set_bnd(n, b, x);
    }
  }

void diffuse(unsigned int n, boundary b, float * x, const float * x0, float diff, float dt)
{
    float a = dt * diff * n * n;
    lin_solve(n, b, x, x0, a, 1 + 4 * a);
}

//float max(float x, float y) {
// return x < y ? y : x;
//}

//float min(float x, float y) {
//  return x < y ? x : y;
//}


__global__ void advect_kernel(unsigned int n, boundary b, float*  d, float* d0, const float* u, const float* v, float dt) {
  float dt0 = dt * n;
  unsigned int i = blockDim.y * blockIdx.y + threadIdx.y + 1;
  unsigned int j = blockDim.x * blockIdx.x + threadIdx.x + 1;
  if (i < n+1 && j < n+1) {
    int i0, i1, j0, j1;
    float x, y, s0, t0, s1, t1;
    x = i - dt0 * u[IX(i, j)];
    y = j - dt0 * v[IX(i, j)];
    x = max(x, 0.5f);
    x = min(x, n + 0.5f);
    i0 = (int)x;
    i1 = i0 + 1;
    y = max(y, 0.5f);
    y = min(y, n + 0.5f);
    j0 = (int)y;
    j1 = j0 + 1;
    s1 = x - i0;
    s0 = 1 - s1;
    t1 = y - j0;
    t0 = 1 - t1;
    d[IX(i, j)] = s0 * (t0 * d0[IX(i0, j0)] + t1 * d0[IX(i0, j1)]) + s1 * (t0 * d0[IX(i1, j0)] + t1 * d0[IX(i1, j1)]);
  }
}

void advect(unsigned int n, boundary b, float*  d, float* d0, const float* u, const float* v, float dt)
{
  unsigned int numBlocks = (n + 31) / 32;
  dim3 block(32, 32);
  dim3 grid(numBlocks,numBlocks);
  advect_kernel<<<grid, block>>>(n, b, d, d0, u, v, dt);
  set_bnd(n, b, d);
}

__global__ void project_density_kernel(unsigned int n, float *u, float *v, float *p, float *div) {
  unsigned int i = blockDim.y * blockIdx.y + threadIdx.y + 1;
  unsigned int j = blockDim.x * blockIdx.x + threadIdx.x + 1;
  if (i < n+1 && j < n+1) {
    div[IX(i, j)] = -0.5f * (u[IX(i + 1, j)] - u[IX(i - 1, j)] +
                            v[IX(i, j + 1)] - v[IX(i, j - 1)]) / n;
    p[IX(i, j)] = 0;
  }
}

__global__ void project_vel_kernel(unsigned int n, float *u, float *v, float *p) {
  unsigned int i = blockDim.y * blockIdx.y + threadIdx.y + 1;
  unsigned int j = blockDim.x * blockIdx.x + threadIdx.x + 1;
  if (i < n+1 && j < n+1) {
    u[IX(i, j)] -= 0.5f * n * (p[IX(i + 1, j)] - p[IX(i - 1, j)]);
    v[IX(i, j)] -= 0.5f * n * (p[IX(i, j + 1)] - p[IX(i, j - 1)]);
  }
}

static void project(unsigned int n, float * u, float *  v, float * p, float * div)
{
        // printf("Thread %d in range [%d,%d), total: %d\n", omp_get_thread_num(), start+1, end+1, n);
    unsigned int numBlocks = (n + 31) / 32;
    dim3 block(32, 32);
    dim3 grid(numBlocks,numBlocks);
    project_density_kernel<<<grid,block>>>(n, u, v, p, div);

    set_bnd(n, NONE, div);
    set_bnd(n, NONE, p);

    lin_solve(n, NONE, p, div, 1, 4);

    project_vel_kernel<<<grid, block>>>(n, u, v, p);
    set_bnd(n, VERTICAL, u);
    set_bnd(n, HORIZONTAL, v);
}

__host__ void dens_step(unsigned int n, float *x, float *x0, float *u, float *v, float diff, float dt)
{
    add_source(n, x, x0, dt);
    SWAP(x0, x);
    diffuse(n, NONE, x, x0, diff, dt);
    SWAP(x0, x);
    advect(n, NONE, x, x0, u, v, dt);
}

__host__ void vel_step(unsigned int n, float *u, float *v, float *u0, float *v0, float visc, float dt)
{
    add_source(n, u, u0, dt);
    add_source(n, v, v0, dt);
    SWAP(u0, u);
    diffuse(n, VERTICAL, u, u0, visc, dt);
    SWAP(v0, v);
    diffuse(n, HORIZONTAL, v, v0, visc, dt);
    project(n, u, v, u0, v0);
    SWAP(u0, u);
    SWAP(v0, v);
    advect(n, VERTICAL, u, u0, u0, v0, dt);
    advect(n, HORIZONTAL, v, v0, u0, v0, dt);
    project(n, u, v, u0, v0);
}
